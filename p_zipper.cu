#include "hip/hip_runtime.h"
/* 
 * PZipper - Merges point clouds and zippers in parallel using CUDA
 *
 * by: Simon Bloch and Martina Costagliola
 * date: 4/7/2016
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>    
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <sys/time.h>
#include <math.h>

#include "lib/eig3.h"
#include "lib/pca.h"
#include "lib/svd.h"
#include "lib/myopengllib.h"

#define printhere printf("HERE %d\n", __LINE__)
#define NUMPOINTS 1
#define HIST_PROPORTION .85
#define CANDIDATES 10

#define DIST_THRESH 0.02


#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}
#define U 7
#define W (U - 1)/2
#define K_POINTS U*U

#define PARTITION 8
#define P (PARTITION*PARTITION)


////////////////////////////////////////////////////////////////////////////////
////////////////////////////// Struct Definitions //////////////////////////////
////////////////////////////////////////////////////////////////////////////////

typedef struct point {   
    int valid;
    float x;
    float y;
    float z;
}pt;

typedef struct candidate{
   float score;
   int i;
   int j;

}candidate;


typedef struct scan_data_state {
  
    pt* base_scan;
    pt* cur_scan;
    pt* cur_normals;
    pt* base_normals;
    float* histograms_A;
    float* histograms_B;
    int* tally_points_A;
    int* tally_points_B;
    candidate* chunk_scores;
}scan_data_state;

scan_data_state scan_data;


///////////////////////////////////////////////////////////////////////////////
///////////////////////////// Function Prototypes //////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//
// global function prototypes

__global__ void generate_histograms(pt* points_list, pt* normals, float* histograms, int* point_pair_tally,
                                    int n, int m, int k, int l);

__global__ void scale_hist(float* histograms, int m, int* tally_points);


__global__ void score_hists(float* cur_hists, float* base_hists, candidate* full_scores,
                            int n, int m, int n2, int m2, int a_ind, int b_ind);

__global__ void rec_sort(candidate* scores, int len, int count);

// device functions prototypes
__device__ void scale_hist(float* histograms, int i, int j, int m, int point_pair_tally);
__device__ float hist_dist(float* hA, float* hB, int i, int j);
__device__ void consider_candidate(float score, int i, int j, candidate* scores);
__device__ int hashFeatures(float* f, float* s);
__device__ pt map(int ind, int i, int j, int n, int m, pt* mat);
__device__ void Drec_sort(candidate* scores, int len, int count);

 __device__ pt Cross(pt A, pt B);
 __device__ pt Sub(pt A, pt B);
 __device__ float Norm(pt A);
 __device__ float Max(float* list, int len);
 __device__ float Dot(pt A, pt B);
 __device__ float Min(float* list, int len);

candidate* generate_scores(scan_data_state scan_data, int n, int m, int n2, int m2);

void generate_hists(scan_data_state scan_data, int n, int m, int n2, int m2);

void scale_histograms(float* histograms, int* tally_points, int n, int m);

void clean_normals(pt* normals, int n, int m);

// prototypes for normal c functions

void write_ply(float** R, pt* pts, char* outfile_name, int n, int m);

float** get_rough_transform(candidate* scores, pt* A, pt* B, int n, int m, int n2, int m2);

pt* run_pca(pt* points, int n, int m, int v);

pt* parse_ply_pts(char* infile_name, float*** vertices, int* n, int* m, int* v);

void Crec_sort(candidate* scores, int len, int count);

void print_mat(float** M);

void merge(candidate* scores, candidate* input);

float CDot(pt A, pt B);
pt CSub(pt A, pt B);
float CNorm(pt A);
pt Cmap(int ind, int i, int j, int n, int m, pt* mat);

////////////////////////////////////////////////////////////////////////////////
//////////////////////////////// Main Function /////////////////////////////////
////////////////////////////////////////////////////////////////////////////////


int main(int argc, char** argv) {


    // grab input files
    char* infile_1_name = argv[1];
    char* infile_2_name = argv[2];
    char* outfile_1_name = argv[3];

    // declare files
    int n, m, v;
    int n2, m2, v2;

    pt* cur_list;
    pt* base_list;
    pt* cur_normals = NULL;
    pt* base_normals = NULL;

    float** vertices;
    //float* histograms;

    // Parse the ply files into an 1D, length (m*n) array
    cur_list = parse_ply_pts(infile_1_name, &vertices, &n, &m, &v);
    base_list = parse_ply_pts(infile_2_name, &vertices, &n2, &m2, &v2);

    if (cur_list==NULL) {
        printf("Cur list is NULL\n");
    }

    if (base_list==NULL) {
        printf("base list is NULL\n");
    }

    // Generate normal vectors at each point
    cur_normals = run_pca(cur_list, n, m, v);
    base_normals = run_pca(base_list, n2, m2, v2);

    // Iterate over normals (sequentially) checking that they all point the same direction
    clean_normals(cur_normals, n, m);
    clean_normals(base_normals, n2, m2);

    /* Malloc on GPU space */
    HANDLE_ERROR(hipMalloc((void **)&scan_data.cur_scan,
                             sizeof(pt) * n * m), "malloc cur_scan");

    HANDLE_ERROR(hipMalloc((void **)&scan_data.base_scan,
                            n2 * m2 * sizeof(pt)), "malloc base_scan");

    HANDLE_ERROR(hipMalloc((void **)&scan_data.base_normals,
                            n2 * m2 * sizeof(pt)), "malloc space for base normal vectors");
    
    HANDLE_ERROR(hipMalloc((void **)&scan_data.cur_normals,
                            n * m * sizeof(pt)), "malloc space for cur normal vectors");
 

    HANDLE_ERROR(hipMalloc((void **)&scan_data.histograms_A,
                                  n * m *16* sizeof(float)), "malloc space for cur hists");
  
    HANDLE_ERROR(hipMalloc((void **)&scan_data.histograms_B,
                                  n2 * m2 *16* sizeof(float)), "malloc space for base hists");

    HANDLE_ERROR(hipMalloc((void **)&scan_data.tally_points_A,
                                  n * m * sizeof(int)), "malloc space for cur hist point tallies");

    HANDLE_ERROR(hipMalloc((void **)&scan_data.tally_points_B,
                                  n2 * m2 * sizeof(int)), "malloc space for base hist point tallies");
    
    /* Copy over to the GPU*/
    HANDLE_ERROR(hipMemcpy(scan_data.cur_scan, cur_list,
                            n * m * sizeof(pt), hipMemcpyHostToDevice),
                 "copy cur points list to GPU");

    HANDLE_ERROR(hipMemcpy(scan_data.base_scan, base_list,
                            n2 * m2 * sizeof(pt), hipMemcpyHostToDevice),
                 "copy base points list to GPU");

    HANDLE_ERROR(hipMemcpy(scan_data.cur_normals, cur_normals,
                                      n * m * sizeof(pt), hipMemcpyHostToDevice),
                         "copy cur normals list to GPU");

    HANDLE_ERROR(hipMemcpy(scan_data.base_normals, base_normals,
                            n2 * m2 * sizeof(pt), hipMemcpyHostToDevice),
                 "copy base normals list to GPU");


    //////////////////////////////////////////////////////
    // PARALLELIZATION BEGINS!!! /////////////////////////
    //////////////////////////////////////////////////////

    hipMemset(scan_data.histograms_A, 0, m * n * 16 * sizeof(float));    
    hipMemset(scan_data.histograms_B, 0, m2 * n2 * 16 * sizeof(float));    
    hipMemset(scan_data.tally_points_A, 0, m * n * sizeof(int));    
    hipMemset(scan_data.tally_points_B, 0, m2 * n2 * sizeof(int));    

    // Generate the histograms for both cur and base scans
    //     (Generate hists populates scan_data with both histograms)
    generate_hists(scan_data, n, m, n2, m2);


    candidate* scores;
    scores = generate_scores(scan_data, n, m, n2, m2);

    // compute the rotation matrix that relates the two scans
    float** R;
    R = get_rough_transform(scores, cur_list, base_list, n, m, n2, m2);

    // write the completed ply files
    write_ply(R, cur_list, outfile_1_name, n, m);

    // free allocated memory on both the CPU and GPU
    free(cur_list);
    free(base_list);
    free(cur_normals);
    free(base_normals);
    free(scores);
    free(R);

    hipFree(scan_data.cur_scan);
    hipFree(scan_data.cur_normals);
    hipFree(scan_data.histograms_A);
    hipFree(scan_data.base_scan);
    hipFree(scan_data.base_normals);
    hipFree(scan_data.histograms_B);
    hipFree(scan_data.chunk_scores);
            
    return 0;
    
}

// Parse the points from each ply file
pt* parse_ply_pts(char* infile_name, float*** vertices, int* n, int* m, int* v) {
    
    FILE *stream;

    if ((stream = fopen(infile_name,"r")) == NULL) {
        fprintf(stderr, "p_zipper: cannot open file %s\n", infile_name);
        fprintf(stderr, "Exiting to system.");
        exit(1);
    }

    //18 then cols then 2 then rows
    char junk_val[20];
    int i;

    // Step thru irrelevant header data til #cols
    for (i = 0; i < 18; i++) {
        fscanf(stream, "%s", junk_val);
    }

    // Read #cols
    fscanf(stream, "%d", m);

    // More junk
    fscanf(stream, "%s", junk_val);
    fscanf(stream, "%s", junk_val);

    // Read #rows
    fscanf(stream, "%d", n);

    // More junk
    for (i = 0; i < 29; i++) {
        fscanf(stream, "%s", junk_val);
    }

    // Read #vertices
    fscanf(stream, "%d", v);

    // Finish reading junk
    for (i = 0; i < 18; i++) {
        fscanf(stream, "%s", junk_val);
    }

    // Now read in data.
    pt *data = NULL;


    // Read in list of vertices
    *vertices = (float**) malloc(*v * sizeof(float *));
    float** v_array = *vertices;

    for (i = 0; i < *v; i++) {

        v_array[i] = (float*) malloc(3 * sizeof(float));

        fscanf(stream, "%f %f %f", &v_array[i][0], &v_array[i][1], &v_array[i][2]);

    }

    int is_vert, index;


    // Read vertices into data depending on whether each point is valid
    data = (pt*) malloc((*n) * (*m) * sizeof(pt));
    for (i = 0; i < (*n) * (*m) ; i++) {
        fscanf(stream, "%d", &is_vert);
        if (is_vert) {
            fscanf(stream, "%d", &index);
            data[i].x = v_array[index][0];
            data[i].y = v_array[index][1];
            data[i].z = v_array[index][2];
                
            free(v_array[index]);

            data[i].valid = 1;         

        } else {

            data[i].valid = 0;
        }
    }
    free(v_array);

    return data;
}

// run PCA on sets of 3D points to aquire normals
pt* run_pca(pt* points, int n, int m, int v) {
    
    int i, j, k, l;
    
    pt* normals;
    normals = (pt*) malloc((n) * (m) * sizeof(pt));

    
    // Declare Kernel
    float** kernel;
    kernel = (float**) malloc(K_POINTS * sizeof(float*));

    int ind;

    float mean[3];
    float V[3][3];
    float d[3];

    for (i=0; i < n; i++) {
        for (j=0; j<m; j++) {
            // AT i, j
            ind = 0;
            
            normals[i*m+j].valid = 1;       
            
            // At i,j loop across kernel and populate with kernel points
            for (k = i - 3; k < i + 4; k++) {
                for (l = j - 3; l < j + 4; l++) {
                    if ((k >= 0) && (k < n) && (l >= 0) && (l < m)) {
                        if (points[k * m + l].valid != 0) {
                            if (CNorm(CSub(points[i*m+j],
                                         points[k*m + l])) < DIST_THRESH) {

                                kernel[ind] = (float*) malloc(3 * sizeof(float));

                                //printf(" points: %f %f %f\n", points[k * m + l].x,
                                //                              points[k * m + l].y,
                                //                              points[k * m + l].z );

                                kernel[ind][0] = points[k * m + l].x;
                                kernel[ind][1] = points[k * m + l].y;
                                kernel[ind][2] = points[k * m + l].z;
                                ind++;
                            }
                        }                        
                    }    
                } // Out of loop across kernel
            }

            // If we have enough points to approximate a plane, run PCA on the kernel points
            // to get normal vector V
            if (ind >= 3) {
                
                PCA(ind, kernel, mean, V, d);

                normals[i*m+j].x = V[0][0];
                normals[i*m+j].y = V[1][0];
                normals[i*m+j].z = V[2][0];
                    
                // printf("i: %d\nj: %d\ne: %d\n", i, j, e);
                // printf("normals[i].x: %f\n", normals[i*m+j].x);                            
                // printf("normals[i].y: %f\n", normals[i*m+j].y);
                // printf("normals[i].z: %f\n", normals[i*m+j].z);
            } else {
                normals[i*m+j].valid = 0;
            }
        }
    }

    return normals;
}

// function to ensure the normals of each scan are all oriented in the same direction
void clean_normals(pt* normals, int n, int m) {

    int cur;

    cur = 0;

    while (!normals[cur].valid) {
        cur++;
    }


    // Loop across vectors
    int i;
    for (i=1; i < n*m; i++) {

        // If the vector is valid
        if (normals[i].valid) {
            if ((cur % m) > (i % m)) {


                // Look at the normal one index up in the matrix
                if (i - m >= 0) {
                    cur = i - m;
                    
                    while (!normals[cur].valid) {
                        cur++;
                    }
                }
            }

            // the dot product with the cur normal is negative, flip this one
            if (CDot(normals[cur], normals[i]) < 0) {

                normals[i].x *= -1;
                normals[i].y *= -1;
                normals[i].z *= -1;

                cur = i;
            }
        }
    }
}

// Populate scan_data with histograms
void generate_hists(scan_data_state scan_data, int n, int m, int n2, int m2){

    // declare error value for checking
    hipError_t error;


    // specify thread and block dimensions
    dim3 blocks(n/PARTITION,m/PARTITION,1);
    dim3 threads_block(PARTITION, PARTITION, 1);

    int k, l;

    // Loop across kernel
    for(k = 0; k < K_POINTS - 1; k++){
        for(l = k + 1; l < K_POINTS; l++){
            error = hipGetLastError();
            if (error != hipSuccess) {
                printf("CudaThread Synchronize Error %s\n", hipGetErrorString(error));
            }

            // Update histograms with the data from kernel index k,l in parallel
            generate_histograms<<<blocks, threads_block>>>(scan_data.cur_scan, scan_data.cur_normals,
                                                           scan_data.histograms_A, scan_data.tally_points_A,
                                                           n, m, k, l);

            // synchronize threads, make sure they are in sync before continuing to the next cuda kernel call
            error = hipDeviceSynchronize();
            if (error != hipSuccess) {
                printhere;
                printf("CudaThread Synchronize Error %s\n", hipGetErrorString(error));
            }
        }
    }

    scale_histograms(scan_data.histograms_A, scan_data.tally_points_A, n, m);
    for(k = 0; k < K_POINTS - 1; k++){
        for(l = k + 1; l < K_POINTS; l++){
            error = hipGetLastError();
            if (error != hipSuccess) {
                printf("CudaThread Synchronize Error %s\n", hipGetErrorString(error));
            }

            // Call generate histograms for scan 2- second cuda kernel call
            generate_histograms<<<blocks, threads_block>>>(scan_data.base_scan, scan_data.base_normals,
                                                           scan_data.histograms_B, scan_data.tally_points_B,
                                                           n2, m2, k, l);

            // synchronize threads, make sure they are in sync before continuing to the next cuda kernel call
            error = hipDeviceSynchronize();
            if (error != hipSuccess) {
                printhere;
                printf("CudaThread Synchronize Error %s\n", hipGetErrorString(error));
            }
        }
    }

    // Use the updated tally_points array to normalize every histogram
    scale_histograms(scan_data.histograms_B, scan_data.tally_points_B, n2, m2);
}

// Normalize every histogram
void scale_histograms(float* histograms, int* tally_points, int n, int m) {
    
    hipError_t error;

    // specify thread and block dimensions
    dim3 blocks(n/PARTITION,m/PARTITION,1);
    dim3 threads_block(PARTITION, PARTITION, 1);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CudaThread Synchronize Error %s\n", hipGetErrorString(error));
    }

    // In parallel, normalize every histogram using the tally points array from scan data
    scale_hist<<<blocks, threads_block>>>(histograms, m, tally_points);

    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        printf("CudaThread Synchronize Error %s\n", hipGetErrorString(error));
    }

}

// Loops across scan A, compares hists with scan B in parallel, accumulates top 10 scores
candidate* generate_scores(scan_data_state scan_data, int n, int m, int n2, int m2) {

    hipError_t error;

    dim3 blocks_2(PARTITION, 1);
    dim3 threads_block_2(PARTITION);

    dim3 blocks_3(1);
    dim3 threads_block_3(1);


    // Thus, idx goes from 0 to P-1

    // Chunk_scores is size 10*P
    HANDLE_ERROR(hipMalloc((void **)&scan_data.chunk_scores,
                            CANDIDATES * P * sizeof(candidate)),
                 "malloc chunk_scores");

    int num_chunks;
    num_chunks = n*m/P;
    
    printf("n: %d\nm: %d\n", n, m);
    printf("NC: %d\n", num_chunks);
    printf("P: %d\n", P);


    candidate* scores;
    scores = (candidate*) malloc(CANDIDATES * sizeof(candidate));

    candidate* input;
    input = (candidate*) malloc(CANDIDATES * sizeof(candidate));

    candidate bad;
    bad.score = 2;

    int i, j;
    for (i=0; i < CANDIDATES; i++) {
        scores[i] = bad;
        input[i] = bad;
    }

    // Loop over chunks of A
    j = 0;
    for (i = 0; i < num_chunks; i++) {

        error = hipGetLastError();
        if (error != hipSuccess) {
            printhere;
            printf("CudaThread Synchronize Error %s\n", hipGetErrorString(error));
        }

        score_hists<<<blocks_2, threads_block_2>>>(scan_data.histograms_A,
                                                   scan_data.histograms_B,
                                                   scan_data.chunk_scores,
                                                   n, m, n2, m2, i, j);
     
        // Now we have 10 * P scores for the first chunk, let's send these up to the CPU
   
        error = hipDeviceSynchronize();

        if (error != hipSuccess) {
            printhere;
            printf("CudaThread Synchronize Error %s\n", hipGetErrorString(error));
        }

            
        // Recursively sort the kernel output scores
        rec_sort<<<blocks_3, threads_block_3>>>(scan_data.chunk_scores, CANDIDATES * P, 0);

        // now do a device to host memcopy so that we can have access to the computed scores
        HANDLE_ERROR(hipMemcpy(input,
                                scan_data.chunk_scores,
                                CANDIDATES * sizeof(candidate),
                                hipMemcpyDeviceToHost),
                     "copy scores to CPU\n");

        // Merge the incoming array of 10 scores with the current top 10
        merge(scores, input);

    }

    free(input);

    return scores;
}


// Generate a rotation matrix from point correspondences
float** get_rough_transform(candidate* scores, pt* A, pt* B, int n, int m, int n2, int m2) {

    int i, j, k, ii, jj;

    float centroidA[3];
    float centroidB[3];

    memset(centroidA, 0, 3*sizeof(float));
    memset(centroidB, 0, 3*sizeof(float));


    pt point, pointA, pointB;
    pointA.valid = 0;
    pointB.valid = 0;

    // Get the centroid of every candidate
    for (i = 0; i < CANDIDATES; i++ ){

        for (j=0; j < K_POINTS; j++) {

            k = scores[i].i/16;

            ii= k/m;
            jj= k%m;

            point = Cmap(j, ii, jj, n, m, A);
            if (point.valid) {
                if (!pointA.valid || j <= (K_POINTS/2)) {
                    pointA = point;
                }
            }

            k = scores[i].j/16;

            ii= k/m;
            jj= k%m;

            point = Cmap(j, ii, jj, n, m, B);
            if (point.valid) {
                if (!pointB.valid || j <= (K_POINTS/2)) {
                    pointB = point;
                }
            }
        }
        printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");

        centroidA[0] += pointA.x;
        centroidA[1] += pointA.y;
        centroidA[2] += pointA.z;

        centroidB[0] += pointB.x;
        centroidB[1] += pointB.y;
        centroidB[2] += pointB.z;

    }
    centroidA[0] /= CANDIDATES;
    centroidA[1] /= CANDIDATES;
    centroidA[2] /= CANDIDATES;
    centroidB[0] /= CANDIDATES;
    centroidB[1] /= CANDIDATES;
    centroidB[2] /= CANDIDATES;

    // H is the covariance matrix of points
    float** H;
    H = (float**)malloc(3*sizeof(float*));
    for (i=0; i < 3; i++) {
        H[i] = (float*)malloc(3*sizeof(float));
        memset(H[i], 0, (3*sizeof(float)));
    }

    float ptA[3];
    float ptB[3];


    // Populate H with covariance values from point correspondences
    for (i=0; i < CANDIDATES; i++) {
        ptA[0] = A[scores[i].i/16].x;
        ptA[1] = A[scores[i].i/16].y;
        ptA[2] = A[scores[i].i/16].z;

        ptB[0] = B[scores[i].j/16].x;
        ptB[1] = B[scores[i].j/16].y;
        ptB[2] = B[scores[i].j/16].z;

        for (j=0; j < 3; j++) {
            for (k=0; k < 3; k++) {
                H[j][k] += (ptA[j] - centroidA[j]) * (ptB[k] - centroidB[k]);
            }
        }
    }
    
    float* w;
    float** v;
    float** R;


    // Malloc space for the SVD Output
    w = (float*)malloc(3*sizeof(float));
    memset(w, 0, 3*sizeof(float));

    v = (float**)malloc(3*sizeof(float*));
    for (i=0; i<3; i++) {
        v[i] = (float*)malloc(3*sizeof(float));
        memset(v[i], 0, 3*sizeof(float));
    }

    R = (float**)malloc(3*sizeof(float*));
    for (i=0; i<3; i++) {
        R[i] = (float*)malloc(3*sizeof(float));
        memset(R[i], 0, 3*sizeof(float));
    }

    int ret;
    printf("H Before\n");
    print_mat(H);


    // RUN SVD To get output matrices
    ret = dsvd(H, 3, 3, w, v);

    printf("H After\n");
    print_mat(H);

    printf("v Matrix\n");
    print_mat(v);

    printf("RET: %d\n", ret);

    //TODO: Replace this with this
    //R[0][0] = 1;
    //R[1][1] = 1;
    //R[2][2] = 1;

    // Perform matrix multiplication to get the output matrix R
    for (i=0; i < 3; i++) {
        for (j=0; j < 3; j++) {
            for (k=0; k < 3; k++) {
                R[i][j] += v[i][k] * H[j][k];
            }
        }
    }

    printf("R Matrix\n");
    print_mat(R);
    return R;
}

// write a ply file given the size of the scan data and the set of points
void write_ply(float** R, pt* pts, char* outfile_name, int n, int m) {
    FILE* out;
    out = fopen(outfile_name, "w");
    
    float p_in[3];
    float p_out[3];

    // Loop over all vertices and print them out
    int i, j, k, ind=0;
    for (i=0; i < n*m; i++) {

        if (pts[i].valid) {
            p_in[0] = pts[i].x;
            p_in[1] = pts[i].y;
            p_in[2] = pts[i].z;

            memset(p_out, 0, 3*sizeof(float));
            for (j=0; j < 3; j++) {
                for (k=0; k < 3; k++) {
                    p_out[j] += R[j][k] * p_in[k];
                }
            }
            
            fprintf(out, "%f %f %f\n", p_out[0], p_out[1], p_out[2]);
            if (p_out[2] > 0.1) {
                pts[i].z -= .2;
            }
        }

    }

    //Print out each point's validity and index
    for (i=0; i < n*m; i++) {
        if (pts[i].valid) {
            fprintf(out, "1 %d\n", ind);
            ind++;
        } else {
            fprintf(out, "0\n");
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////// Kernel Functions ///////////////////////////////
////////////////////////////////////////////////////////////////////////////////


// CUDA kernel function call to generate histograms for a given scan
__global__ void generate_histograms(pt* points_list, pt* normals, float* histograms, int* point_pair_tally,
                                    int n, int m, int k, int l) {

    int ind;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    //int offset = idx + idy*m;
    
    // 
    // Dependent on indices
    pt pt_k;
    pt pt_l;
    pt normal_k;
    pt normal_l;

    // Independent of indices
    pt pt_s;
    pt pt_t;
    pt normal_s;
    pt normal_t;
    pt u;
    pt v;
    pt w;

    float f[4], s[4], distances[4];

    //for (i=0; i < some x row value; i++ )
    //  for (i=0; i < some y column dimension for the block; j++)

    // AT i, j
    memset(distances, 0, 4*sizeof(float));


    // Get the radius of the kernel's edge points
    if ((idx > W) && (idy > W)) {
        distances[0] = Norm(Sub(points_list[(idx*m) + idy], points_list[(idx-W)*m + (idy-W)]));
    }
    if ((idx > W) && (idy + W < m)) {
        distances[1] = Norm(Sub(points_list[(idx*m) + idy], points_list[(idx-W)*m + (idy+W)]));
    }
    if ((idx + W < n) && (idy > W)) {
        distances[2] = Norm(Sub(points_list[(idx*m) + idy], points_list[(idx+W)*m + (idy-W)]));
    }
    if ((idx + W < n) && (idy + W < m)) {
        distances[3] = Norm(Sub(points_list[(idx*m) + idy], points_list[(idx+W)*m + (idy+W)])); 
    }
    
    // Compute kernel radius from edge points of kernel
    s[1] = Max(distances, 4);

    /*
      We need neighboring points, so we can index into points list
      we need neighboring normals, so we can index into normals list
    */
    /* for (k = 0; k < K_POINTS - 1; k++) { */
    /*     for (l = k + 1; l < K_POINTS; l++) { */

    // Get point pair and normal pair data
    pt_k = map(k, idx, idy, n, m, points_list);
    pt_l = map(l, idx, idy, n, m, points_list);
    normal_k = map(k, idx, idy, n, m, normals);
    normal_l = map(l, idx, idy, n, m, normals);

    if (pt_k.valid && pt_l.valid) {

        //Cross Dot Sub
        if (Dot(normal_k, Sub(pt_l, pt_k)) <=
            Dot(normal_l, Sub(pt_k, pt_l))) {
                        
            pt_s = pt_k;
            pt_t = pt_l;
            normal_s = normal_k;
            normal_t = normal_l;
        } else {
            pt_s = pt_l;
            pt_t = pt_k;
            normal_s = normal_l;
            normal_t = normal_k;
        }
        u = normal_s;
        v = Cross(Sub(pt_t, pt_s), u);
        w = Cross(u, v);

        f[0] = Dot(v, normal_t);
        f[1] = Norm(Sub(pt_t, pt_s));
        f[2] = Dot(u, Sub(pt_t, pt_s)) / f[2];
        f[3] = atan2(Dot(w, normal_t), Dot(u, normal_t));


        ind = hashFeatures(f, s);
        if ((ind < 0) || (ind > 15)) {
            printf("EXITING: IMPROPER HASH\n");
            //exit(1);
        }

        // Update the histograms with the hashed features digit
        histograms[((idx*m + idy) * 16) + ind]++;
        point_pair_tally[(idx*m) + idy]++;
    }

}

// CUDA kernel function call which uses the tally_points array to scale each histogram
// in a given scan
__global__ void scale_hist(float* histograms, int m, int* tally_points) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int possibles = K_POINTS * (K_POINTS - 1)/2;
    
    int tally_threshold = possibles * HIST_PROPORTION;
        
    int p;

    if (tally_points[i*m + j] < tally_threshold) {
        histograms[(i*m + j)*16] = -1;
    } else {
        for (p=(i*m+j)*16; p < (i*m+j)*16 + 16; p++) {
            //printf("%f ", histograms[p]);
            histograms[p] /= tally_points[i*m + j];
        }
    }
}



// CUDA kernel function call to score given histograms
__global__ void score_hists(float* cur_hists, float* base_hists,
                            candidate* chunk_scores,
                            int n, int m, int n2, int m2,
                            int a_ind, int b_ind) {


    
    // index appropriately into the correct block and thread within the grid
    int i, j, k;

    // Goes from 0 to P-1
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);

    i = 16 * (a_ind * P + idx);
    
    float score;

    // Initialize all scores to the max
    for(k = idx * CANDIDATES;
        k < idx * CANDIDATES + CANDIDATES;
        k++) {
    
        chunk_scores[k].score = 2;
    }

    // Loop across scan B, get scores
    for (j = 0;
         j < n*m*16;
         j+=16) {

        // compute the score between two histograms
        score = hist_dist(cur_hists, base_hists, i, j);

        consider_candidate(score, i, j,
                           &chunk_scores[idx * CANDIDATES]);
    }
}

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////// Helper Functions ///////////////////////////////
////////////////////////////////////////////////////////////////////////////////

// Merge the scores from two lists of size 10 to get the top 10
void merge(candidate* scores, candidate* input) {

    int cur_1 = 0;
    int cur_2 = 0;

    candidate temp;

    while ((cur_1 < CANDIDATES) && (cur_2 < CANDIDATES)) {
        if (input[cur_2].score > scores[cur_1].score) {
            temp = scores[cur_1];
            scores[cur_1] = input[cur_2];
            input[cur_2] = temp;
            cur_1++;

        } else {
            cur_2++;
        }
    }
}

// Recursive sort on the CPU
void Crec_sort(candidate* scores, int len, int count) {

    if (count == CANDIDATES) {
        return;
    }

    int best, i;
    best = 0;

    for (i=1; i<len; i++) {
        if (scores[best].score < scores[i].score) {
            best = i;
        }
    }

    candidate temp;

    temp = scores[0];
    scores[0] = scores[best];
    scores[best] = temp;

    Crec_sort(&(scores[1]), len - 1, count + 1);
}

// Recursive sort on the GPU
__global__ void rec_sort(candidate* scores, int len, int count) {

    if (count == CANDIDATES) {
        return;
    }

    int best, i;
    best = 0;

    // Find best score
    for (i=1; i<len; i++) {
        if (scores[best].score < scores[i].score) {
            best = i;
        }
    }
    
    

    candidate temp;

    // Swap up the best score to the top
    temp = scores[0];
    scores[0] = scores[best];
    scores[best] = temp;
    
    Drec_sort(&(scores[1]), len - 1, count + 1);
}

// Device function for recursive sort
__device__ void Drec_sort(candidate* scores, int len, int count) {

    if (count == CANDIDATES) {
        return;
    }

    int best, i;
    best = 0;

    for (i=1; i<len; i++) {
        if (scores[best].score < scores[i].score) {
            best = i;
        }
    }

    candidate temp;

    temp = scores[0];
    scores[0] = scores[best];
    scores[best] = temp;
    
    

    Drec_sort(&(scores[1]), len - 1, count + 1);
}

// Print a matrix
void print_mat(float** M) {
    int i;
    int j;

    printf("+ + + + + + + + + \n");
    printf("MATRIX:\n");
    for (i=0; i<3; i++) {
        for (j=0; j<3; j++) {
            printf("%f ", M[i][j]);
        }
        printf("\n");
    }
    printf("- - - - - - - - - \n\n\n");

}

// Score two histogram distributions
__device__ float hist_dist(float* hA, float* hB, int i, int j) {
    int k;
    float sum;

    if ((hA[i] < 0) || (hB[j] < 0)) {
        return 2;
    }

    sum = 0;
    for (k=0; k<16; k++) {
        sum += (hA[i + k] - hB[j + k]) * (hA[i + k] - hB[j + k]);
    }
    
    return sqrt(sum);
}

// Dot product
float CDot(pt A, pt B) {
    return (A.x * B.x) + (A.y * B.y) + (A.z * B.z);
}

// Norm of a vector	
float CNorm(pt A) {
    return sqrt((A.x * A.x) + (A.y * A.y) + (A.z * A.z));
}

// Difference between vectors
pt CSub(pt A, pt B) {

    B.x = A.x - B.x;
    B.y = A.y - B.y;
    B.z = A.z - B.z;

    return B;
}

// Map a kernel index to a scan point
pt Cmap(int ind, int i, int j, int n, int m, pt* mat) {

    pt invalid;
    invalid.valid = 0;

    // Have: 
    //   ind
    //   U (width of kernel)
    //   W (width of side of kernel)
    //   NB/2 (num candidate neighbors, not nec. valid)
    //   n & m
    int a, b;

    a = ind / U;
    b = ind % U;

    // a, b -- Now span from 0 to (width of kernel - 1)
    // now we need to map a and b to i,j indexes
    // 
    // if a is 0, we want i - W
    // if b is 0, we want j - W
    a = i - W + a;
    b = j - W + b;

    if ((a < 0) ||
        (b < 0) ||
        (a >= n) ||
        (b >= m)) {
        return invalid;
    }

    return mat[a*n + b];
}

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////// Device Functions ///////////////////////////////
////////////////////////////////////////////////////////////////////////////////

// Maps index to its corresponding vertex/normal
__device__ pt map(int ind, int i, int j, int n, int m, pt* mat) {

    pt invalid;
    invalid.valid = 0;

    // Have: 
    //   ind
    //   U (width of kernel)
    //   W (width of side of kernel)
    //   NB/2 (num candidate neighbors, not nec. valid)
    //   n & m
    int a, b;

    a = ind / U;
    b = ind % U;

    // a, b -- Now span from 0 to (width of kernel - 1)
    // now we need to map a and b to i,j indexes
    // 
    // if a is 0, we want i - W
    // if b is 0, we want j - W
    a = i - W + a;
    b = j - W + b;

    if ((a < 0) ||
        (b < 0) ||
        (a >= n) ||
        (b >= m)) {
        return invalid;
    }

    return mat[a*n + b];
}

// Dot product
__device__ float Dot(pt A, pt B) {
    return (A.x * B.x) + (A.y * B.y) + (A.z * B.z);
}


// Get a vector cross product
__device__ pt Cross(pt A, pt B) {
    pt C;
    C.valid = 1;
    C.x = A.y * B.z - A.z * B.y;
    C.y = A.z * B.x - A.x * B.z;
    C.z = A.x * B.y - A.y * B.x;

    return C;
}

// Get a difference between vectors
__device__ pt Sub(pt A, pt B) {

    B.x = A.x - B.x;
    B.y = A.y - B.y;
    B.z = A.z - B.z;

    return B;
}

// Get a vector length
__device__ float Norm(pt A) {
    return sqrt((A.x * A.x) + (A.y * A.y) + (A.z * A.z));
}

// Get the max of a list
__device__ float Max(float* list, int len) {
    
    int i, max = 0;
    for (i = 1; i < len; ++i){
        if (list[i] > list[max]){
            max = i;            
        }
    }
    return list[max];
}

// Get the min of a list
__device__ float Min(float* list, int len) {

      int i, min = 0;
          for (i = 1; i < len; ++i){
              if (list[i] < list[min]){
                    min = i;
                                          }
                        }
              return min;
}

// Hash features into bits using threshold vector s
__device__ int hashFeatures(float* f, float* s) {

    int i, idx = 0;
    for (i = 0; i < 4; i++){
        if (!(f[i] < s[i])){
            idx += pow(2.0,i);
        }
    }

    return idx;
}

// Add a candidate to our scores array if its score is sufficiently high
__device__ void consider_candidate(float score, int i, int j, candidate* scores){
    int cur_ind;
    candidate temp;
    candidate cur;
    cur.score = score;
    cur.i = i;
    cur.j = j;
    if(cur.score > scores[CANDIDATES-1].score){
        cur_ind = CANDIDATES-1;
        scores[cur_ind] = cur;
        while(scores[cur_ind].score > scores[cur_ind -1].score){      
            temp = scores[cur_ind];
            scores[cur_ind] = scores[cur_ind-1];
            scores[cur_ind-1] = temp;
            if(cur_ind == 1){
                break;
            }
            
        }
    }
}

